#include "hip/hip_runtime.h"
﻿// RasterKernel.cu
#include <hip/hip_runtime.h>
#include <>
#include "gpu/RasterKernel.h"

// Raster Helpers
__device__ float edgeFn(const float2 a, const float2 b, const float2 p) {
    return (b.x - a.x) * (p.y - a.y) - (b.y - a.y) * (p.x - a.x);
}

__device__ bool insideTriangle(
    const float2 s0, const float2 s1, const float2 s2,
    const float2 p, float& alpha, float& beta, float& gamma)
{
    float area = edgeFn(s0, s1, s2);
    alpha = edgeFn(p, s1, s2) / area;
    beta = edgeFn(p, s2, s0) / area;
    gamma = edgeFn(p, s0, s1) / area;
    return (alpha >= 0 && beta >= 0 && gamma >= 0);
}

__device__ float interpDepth(
    float z0, float z1, float z2,
    float alpha, float beta, float gamma)
{
    return alpha * z0 + beta * z1 + gamma * z2;
}

__device__ uchar3 interpColorPC(
    const uchar3 c0, const uchar3 c1, const uchar3 c2,
    const float invW0, const float invW1, const float invW2,
    float alpha, float beta, float gamma)
{
    float oW = alpha * invW0 + beta * invW1 + gamma * invW2;
    float r = (alpha * c0.x * invW0 + beta * c1.x * invW1 + gamma * c2.x * invW2) / oW;
    float g = (alpha * c0.y * invW0 + beta * c1.y * invW1 + gamma * c2.y * invW2) / oW;
    float b = (alpha * c0.z * invW0 + beta * c1.z * invW1 + gamma * c2.z * invW2) / oW;
    return make_uchar3(r, g, b);
}

// In CUDA, everytime we launch a kernel with "myKernel<<<gridDim, blockDim>>>(…);"
// the CUDA runtime magically creates a two-level exectution space: Grid of thread 
// blocks, each block of threads. CUDA gives us three built-in integer vectors in 
// every kernel. blockIdx: the 3D index of *this* block within the grid, blockDim:
// the dimension (x,y,z) of *every* block, threadIdx: the 3D index of *this* thread
// within its block. Block size: each block is 16×16 threads, Grid size: enough blocks 
// so that grid.x*16 >= W and grid.y*16 => H. Each thread is responsible for is own pixel 
// calculations. So, once we launch the kernel and the grid is made, any work we need to do 
// on a given pixel (check, rasterize, ...) is executed in parallel with all other pixels.
// i.e. each thread runs this code at the same time. Read below...

// ---------------------------------------------------
// 1) clearBuffersKernel
// ---------------------------------------------------

// Parameters:
// __global__: Marks this as a kernel, i.e. a function that runs on the GPU
//  and is launched from the host via <<<...>>>. 
// uchar3* colorBuf: A pointer into GPU memory for the 2D color buffer, 
// laid out as a flat array of(R, G, B) bytes per pixel.
// float* depthBuff: A pointer into GPU memory for the 2D depth buffer, 
// one float per pixel.
// int W, H: The width and height of the fb.
// uchar3 bgColor: A packed(r, g, b) value for the background.
// float initDepth: The "infinite" depth value(we use FLT_MAX) for clearing
// Note that the pointer to buffers here are gpu sided for now.
//
// Thread indexing: 
// blockDim is the dimensions of each block (here dim3(16,16)), so blockDim.x == 16.
// blockIdx is the coordinates of this block in the 2D grid we launched.
// threadIdx is the coordinates of this thread within its block.
// Multiplying out gives each thread a unique (x,y) in the full image. e.g. x pos of 
// 6th block, 20th thread: so we are at the idx = 5th 16x16 block so thats, 5 * 16 x
// entries along. Then we just add the x coordinate of the thread we want xIdx = 
// 5 * 16 + 3 (3 since we are in the next row but 4 along) = column 83. Same for y.
//
// Bounds check: self explanatory
//
// Buffer indexing:
// Each thread is responsible for is own pixel calculations. They essentially each 
// execute this code block once. It picks out its associted pixel via the blockIdx,
// blockDim, and threadIdx which we've loaded as a buffer index. Then it just does 
// colorBuf[idx] = bgColor; depthBuf[idx] = initDepth; In this case, just fills the 
// associted pixel in the buffer with default values.
__global__ void clearBuffersKernel(
    uchar3* d_colorBuf,
    float* d_depthBuf,
    int     W,
    int     H,
    uchar3  bgColor,
    float   initDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;
    int idx = y * W + x;
    d_colorBuf[idx] = bgColor;
    d_depthBuf[idx] = initDepth;
}

// dim3: a struct that stores 3 dimension, default z = 1. We create
// a 16x16 "block" of threads. Then, we create a (width + 15)/16 x 
// (height + 15)/16 "grid" of blocks. We need enough blocks so that 
// grid.x * block.x >= width and similarly in Y. 
// (width + block.x - 1) / block.x is the standard integer “round up” trick. 
// E.g. if width=1200, (1200 + 15) / 16 = 76 blocks in X, each block covering 
// 16 pixels -> 76×16 = 1216 threads, of which the extra 16 simply do nothing 
// (we guard them in the kernel)
//
// <<<grid, block>>>: special CUDA launch operator recognized by nvcc. It tells 
// the driver: "please run grid.x * grid.y blocks, each with block.x * block.y 
// threads."
//
// hipDeviceSynchronize(): makes the CPU ignore until all previously issued GPU work 
// is completed
void launchClearBuffers(
    uchar3* d_colorBuf,
    float* d_depthBuf,
    int     width,
    int     height)
{
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    clearBuffersKernel <<<grid, block>>> (d_colorBuf, d_depthBuf, 
        width, height, make_uchar3(150, 150, 150), FLT_MAX);
    hipDeviceSynchronize();
}

// Everything is the same CUDA-wise. We are just executing this code 
// on every thread, but this time filling the device frame buffer with
// rasterized pixels.

// ---------------------------------------------------
// 2) rasterKernel stub
// ---------------------------------------------------

// Rasterization
// For every pixel, concurrently, we test only the triangles touching that pixel's
// tile, then interpolate depth and color, and lastly write the closest fragments.
// For a pixel at (x,y): We start with a bounds check and skip if oustside. Next, 
// we do a tile lookup by dividing the pixel coordinate by tilesize and defining 
// that tile. This is the tile that pixel (x,y) lives in. Now we loop through each 
// triangle overlapping the given tile. We fetch its vertices' screen position as well 
// as the precopmuted clip-space interpolation data. Then we compute barycentrics, 
// interpolate depth, test depth, and write like usual.
static __global__
void rasterPixelsKernel(const DevicePrimitive* prims, int numPrims,
    const int* cellOffsets, const int* cellTriIndices,
        uchar3* outColor, float* outDepth, 
            int W, int H,
                int tileSize, int numTilesX,
                    uchar3 bgColor, float initDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    int idx = y * W + x;
    // Seed with clear values
    float bestDepth = outDepth[idx];
    uchar3 bestColor = outColor[idx];

    // Determine tile containing pixel (x,y) and its triangle list
    int tx = x / tileSize;
    int ty = y / tileSize;
    int cell = ty * numTilesX + tx;     // Which tile pixel (x,y) lives in
    int start = cellOffsets[cell];      // The list of triangle IDs that might cover any pixel in cell
    int end = cellOffsets[cell + 1];

    // Test point
    float2 p = make_float2(x + 0.5f, y + 0.5f);

    // Loop ONLY the candidate triangles overlapping this tile (cell)
    for (int ptr = start; ptr < end; ++ptr) {
        int triID = cellTriIndices[ptr];
        const DevicePrimitive& tri = prims[triID];       // The triangle overlapping this tile

        // Screen-space data temp holders
        float2 s[3];            // x,y position of 3 vertices in screen-space
        float  zs[3], invW[3];  // z depth and inverse of ws for 3 vertices
        uchar3 col[3];          // Colors atribured to each vertex

        // Looping through each vertex of test triangle to determine screen-space data
        for (int v = 0; v < 3; ++v) {
            float4 P = tri.clipPos[v];      // Clip-space point vertex
            float wInv = 1.0f / P.w;        // 1/w
            invW[v] = wInv;                 

            // NDC
            float ndcX = P.x * wInv;        // Perspecive divides to NDC             
            float ndcY = P.y * wInv;            
            zs[v] = P.z * wInv;             // post-divide depth

            // toScreen
            s[v].x = (ndcX * 0.5f + 0.5f) * W;
            s[v].y = (1.0f - (ndcY * 0.5f + 0.5f)) * H;

            col[v] = tri.color[v];
        }

        // Inside-triangle test + barycentrics
        float alpha, beta, gamma;
        if (!insideTriangle(s[0], s[1], s[2], p, alpha, beta, gamma)) continue;

        // Depth interpolation and test
        float d = interpDepth(zs[0], zs[1], zs[2], alpha, beta, gamma);
        if (d < bestDepth) {
            bestDepth = d;
            bestColor = interpColorPC(col[0], col[1], col[2], invW[0], invW[1], invW[2], alpha, beta, gamma);
        }
    }

    // 5) write out
    outDepth[idx] = bestDepth;
    outColor[idx] = bestColor;
}

// Launcher that passes clear params and calls kernel
void launchRasterPixels(
    const DevicePrimitive* d_prims,
    int                    numPrims,
    const int* d_cellOffsets,
    const int* d_cellTriIndices,
    uchar3* d_colorBuf,
    float* d_depthBuf,
    int                    width,
    int                    height,
    int                    tileSize,
    int                    numTilesX)
{
    // Build thread grid
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    uchar3 bg = make_uchar3(150, 150, 150);
    float  id = FLT_MAX;
    
    // Call rasterization kernel
    rasterPixelsKernel <<<grid, block >>> (d_prims, numPrims, d_cellOffsets, 
        d_cellTriIndices, d_colorBuf, d_depthBuf, width, height, tileSize, numTilesX,
            bg, id);

    // Tell CPU to wait
    hipDeviceSynchronize();
}