#include "hip/hip_runtime.h"
﻿// RasterKernel.cu
#include <hip/hip_runtime.h>
#include <>
#include <cfloat>              // for FLT_MAX
#include "gpu/RasterKernel.h"

// In CUDA, everytime we launch a kernel with "myKernel<<<gridDim, blockDim>>>(…);"
// the CUDA runtime magically creates a two-level exectution space: Grid of thread 
// blocks, each block of threads. CUDA gives us three built-in integer vectors in 
// every kernel. blockIdx: the 3D index of *this* block within the grid, blockDim:
// the dimension (x,y,z) of *every* block, threadIdx: the 3D index of *this* thread
// within its block. Block size: each block is 16×16 threads, Grid size: enough blocks 
// so that grid.x*16 >= W and grid.y*16 => H. Each thread is responsible for is own pixel 
// calculations. So, once we launch the kernel and the grid is made, any work we need to do 
// on a given pixel (check, rasterize, ...) is executed in parallel with all other pixels.
// i.e. each thread runs this code at the same time. Read below...

// ---------------------------------------------------
// 1) clearBuffersKernel
// ---------------------------------------------------

// Parameters:
// __global__: Marks this as a kernel, i.e. a function that runs on the GPU
//  and is launched from the host via <<<...>>>. 
// uchar3* colorBuf: A pointer into GPU memory for the 2D color buffer, 
// laid out as a flat array of(R, G, B) bytes per pixel.
// float* depthBuff: A pointer into GPU memory for the 2D depth buffer, 
// one float per pixel.
// int W, H: The width and height of the fb.
// uchar3 bgColor: A packed(r, g, b) value for the background.
// float initDepth: The "infinite" depth value(we use FLT_MAX) for clearing
// Note that the pointer to buffers here are gpu sided for now.
//
// Thread indexing: 
// blockDim is the dimensions of each block (here dim3(16,16)), so blockDim.x == 16.
// blockIdx is the coordinates of this block in the 2D grid we launched.
// threadIdx is the coordinates of this thread within its block.
// Multiplying out gives each thread a unique (x,y) in the full image. e.g. x pos of 
// 6th block, 20th thread: so we are at the idx = 5th 16x16 block so thats, 5 * 16 x
// entries along. Then we just add the x coordinate of the thread we want xIdx = 
// 5 * 16 + 3 (3 since we are in the next row but 4 along) = column 83. Same for y.
//
// Bounds check
//
// Buffer indexing:
// Each thread is responsible for is own pixel calculations. They essentially each 
// execute this code block once. It picks out its associted pixel via the blockIdx,
// blockDim, and threadIdx which we've loaded as a buffer index. Then it just does 
// colorBuf[idx] = bgColor; depthBuf[idx] = initDepth; In this case, just fills the 
// associted pixel in the buffer with default values.
__global__ void clearBuffersKernel(
    uchar3* colorBuf,
    float* depthBuf,
    int     W,
    int     H,
    uchar3  bgColor,
    float   initDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;
    int idx = y * W + x;
    colorBuf[idx] = bgColor;
    depthBuf[idx] = initDepth;
}

// dim3: a struct that stores 3 dimension, default z = 1. We create
// a 16x16 "block" of threads. Then, we create a (width + 15)/16 x 
// (height + 15)/16 "grid" of blocks. We need enough blocks so that 
// grid.x * block.x >= width and similarly in Y. 
// (width + block.x - 1) / block.x is the standard integer “round up” trick. 
// E.g. if width=1200, (1200 + 15) / 16 = 76 blocks in X, each block covering 
// 16 pixels -> 76×16 = 1216 threads, of which the extra 16 simply do nothing 
// (we guard them in the kernel)
//
// <<<grid, block>>>: special CUDA launch operator recognized by nvcc. It tells 
// the driver: "please run grid.x * grid.y blocks, each with block.x * block.y 
// threads."
//
// hipDeviceSynchronize(): makes the CPU block until all previously issued GPU work 
// is completed
void launchClearBuffers(
    uchar3* d_colorBuf,
    float* d_depthBuf,
    int     width,
    int     height)
{
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    clearBuffersKernel <<<grid, block>>> (d_colorBuf, d_depthBuf, 
        width, height, make_uchar3(150, 150, 150), FLT_MAX);
    hipDeviceSynchronize();
}

// Everything is the same CUDA-wise. We are just executing this code 
// on every thread, but this time filling the device frame buffer with
// rasterized pixels.

// ---------------------------------------------------
// 2) rasterKernel stub
// ---------------------------------------------------
// edge function (2D cross-product) on the device
__device__ float edgeFn2D(float ax, float ay,
    float bx, float by,
    float px, float py)
{
    return (bx - ax) * (py - ay)
        - (by - ay) * (px - ax);
}

__global__ void rasterKernel(
    const DevicePrimitive* prims,
    int      numPrims,
    uchar3* colorBuf,
    float* depthBuf,
    int      W,
    int      H)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;
    int idx = y * W + x;

    // start with the cleared values
    float bestZ = depthBuf[idx];
    uchar3 bestC = colorBuf[idx];

    // center of this pixel in screen coords
    float px = x + 0.5f, py = y + 0.5f;

    for (int i = 0; i < numPrims; ++i)
    {
        // load clip-space verts
        float4 c0 = prims[i].clipPos[0];
        float4 c1 = prims[i].clipPos[1];
        float4 c2 = prims[i].clipPos[2];

        // 1) perspective divide -> NDC
        float invW0 = 1.0f / c0.w;
        float invW1 = 1.0f / c1.w;
        float invW2 = 1.0f / c2.w;

        float3 ndc0 = make_float3(c0.x * invW0,
            c0.y * invW0,
            c0.z * invW0);
        float3 ndc1 = make_float3(c1.x * invW1,
            c1.y * invW1,
            c1.z * invW1);
        float3 ndc2 = make_float3(c2.x * invW2,
            c2.y * invW2,
            c2.z * invW2);

        // 2) NDC -> screen‐space
        float sx0 = (ndc0.x * 0.5f + 0.5f) * W;
        float sy0 = (1.0f - (ndc0.y * 0.5f + 0.5f)) * H;
        float sz0 = ndc0.z;

        float sx1 = (ndc1.x * 0.5f + 0.5f) * W;
        float sy1 = (1.0f - (ndc1.y * 0.5f + 0.5f)) * H;
        float sz1 = ndc1.z;

        float sx2 = (ndc2.x * 0.5f + 0.5f) * W;
        float sy2 = (1.0f - (ndc2.y * 0.5f + 0.5f)) * H;
        float sz2 = ndc2.z;

        // 3) premultiplied colors
        uchar3 col0 = prims[i].color[0];
        uchar3 col1 = prims[i].color[1];
        uchar3 col2 = prims[i].color[2];

        float r0 = col0.x * invW0, g0 = col0.y * invW0, b0 = col0.z * invW0;
        float r1 = col1.x * invW1, g1 = col1.y * invW1, b1 = col1.z * invW1;
        float r2 = col2.x * invW2, g2 = col2.y * invW2, b2 = col2.z * invW2;

        // 4) compute barycentrics
        float area = edgeFn2D(sx0, sy0, sx1, sy1, sx2, sy2);
        if (area == 0.0f) continue;

        float alpha = edgeFn2D(px, py, sx1, sy1, sx2, sy2) / area;
        float beta = edgeFn2D(px, py, sx2, sy2, sx0, sy0) / area;
        float gamma = 1.0f - alpha - beta;

        // 5) inside‐triangle test
        if (alpha < 0 || beta < 0 || gamma < 0) continue;

        // 6) perspective‐correct depth & color
        float oneOverW = alpha * invW0 + beta * invW1 + gamma * invW2;

        float z = (alpha * sz0 + beta * sz1 + gamma * sz2);

        // depth test (z already in NDC‐space, or we can divide by oneOverW)
        if (z < bestZ)
        {
            // interpolate color
            float r = (alpha * r0 + beta * r1 + gamma * r2) / oneOverW;
            float g = (alpha * g0 + beta * g1 + gamma * g2) / oneOverW;
            float b = (alpha * b0 + beta * b1 + gamma * b2) / oneOverW;

            bestZ = z;
            bestC = make_uchar3((uint8_t)r,
                (uint8_t)g,
                (uint8_t)b);
        }
    }

    // write out
    depthBuf[idx] = bestZ;
    colorBuf[idx] = bestC;
}

void launchRasterKernel(
    const DevicePrimitive* d_prims,
    int numPrims,
    uchar3* d_colorBuf,
    float* d_depthBuf,
    int     width,
    int     height)
{
    dim3 block(16, 16), grid((width + 15) / 16, (height + 15) / 16);
    rasterKernel <<<grid, block>>> (d_prims, numPrims,
        d_colorBuf, d_depthBuf,
        width, height);
    hipDeviceSynchronize();
}
